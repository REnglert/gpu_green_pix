#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include <unistd.h>
#include "string.h"
#include "sobel.h"

#define DEFAULT_THRESHOLD 12000

#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 32

unsigned int *read_ppm( char *filename, int & xsize, int & ysize, int & maxval ){
  
    if ( !filename || filename[0] == '\0') {
        fprintf(stderr, "read_ppm but no file name\n");
        return NULL;  // fail
    }

    // fprintf(stderr, "read_ppm( %s )\n", filename);
    int fd = open( filename, O_RDONLY);
    if (fd == -1){
        fprintf(stderr, "read_ppm()    ERROR  file '%s' cannot be opened for reading\n", filename);
        return NULL; // fail 
    }

    char chars[1024];
    int num = read(fd, chars, 1000);

      if (chars[0] != 'P' || chars[1] != '6'){
        fprintf(stderr, "Texture::Texture()    ERROR  file '%s' does not start with \"P6\"  I am expecting a binary PPM file\n", filename);
        return NULL;
    }

     unsigned int width, height, maxvalue;


    char *ptr = chars+3; // P 6 newline
    if (*ptr == '#'){ // comment line! 
        ptr = 1 + strstr(ptr, "\n");
    }

    num = sscanf(ptr, "%d\n%d\n%d",  &width, &height, &maxvalue);
    // fprintf(stderr, "read %d things   width %d  height %d  maxval %d\n", num, width, height, maxvalue);  
    xsize = width;
    ysize = height;
    maxval = maxvalue;
  
    unsigned int *pic = (unsigned int *)malloc( width * height * sizeof(unsigned int) * 3);
    if (!pic) {
        fprintf(stderr, "read_ppm()  unable to allocate %d x %d unsigned ints for the picture\n", width, height);
        return NULL; // fail but return
    }

  // allocate buffer to read the rest of the file into
    int bufsize =  3 * width * height * sizeof(unsigned char);
    if (maxval > 255) bufsize *= 2;
    unsigned char *buf = (unsigned char *)malloc( bufsize );
    if (!buf) {
        fprintf(stderr, "read_ppm()  unable to allocate %d bytes of read buffer\n", bufsize);
        return NULL; // fail but return
    }

    // TODO really read
    char duh[80];
    char *line = chars;

    // find the start of the pixel data.   no doubt stupid
    sprintf(duh, "%d\0", xsize);
    line = strstr(line, duh);
    //fprintf(stderr, "%s found at offset %d\n", duh, line-chars);
    line += strlen(duh) + 1;

    sprintf(duh, "%d\0", ysize);
    line = strstr(line, duh);
    //fprintf(stderr, "%s found at offset %d\n", duh, line-chars);
    line += strlen(duh) + 1;

    sprintf(duh, "%d\0", maxval);
    line = strstr(line, duh);


    // fprintf(stderr, "%s found at offset %d\n", duh, line - chars);
    line += strlen(duh) + 1;

    long offset = line - chars;
    lseek(fd, offset, SEEK_SET); // move to the correct offset
    long numread = read(fd, buf, bufsize);
    // fprintf(stderr, "Texture %s   read %ld of %ld bytes\n", filename, numread, bufsize); 

    close(fd);

    int pixels = xsize * ysize * 3;
    for (int i=0; i<pixels; i++) pic[i] = (int) buf[i]; 

    return pic; // success
}

void write_ppm( const char *filename, int xsize, int ysize, int maxval, int *pic) 
{
    FILE *fp;
    
    fp = fopen(filename, "w");
    if (!fp) {
            fprintf(stderr, "FAILED TO OPEN FILE '%s' for writing\n");
            exit(-1); 
    }
    
    fprintf(fp, "P6\n"); 
    fprintf(fp,"%d %d\n%d\n", xsize, ysize, maxval);
    
    int numpix = xsize * ysize * 3;
    for (int i=0; i<numpix; i+=3) {
        fprintf(fp, "%c%c%c", (unsigned char) pic[i], (unsigned char) pic[i+1], (unsigned char) pic[i+2]); 
    }
    fclose(fp);
}

void write_ppm_from_bools( const char *filename, int xsize, int ysize, int maxval, bool *pic) 
{
    FILE *fp;
    
    fp = fopen(filename, "w");
    if (!fp) {
            fprintf(stderr, "FAILED TO OPEN FILE '%s' for writing\n");
            exit(-1); 
    }
    
    fprintf(fp, "P6\n"); 
    fprintf(fp,"%d %d\n%d\n", xsize, ysize, maxval);
    
    int numpix = xsize * ysize;
    for (int i=0; i<numpix; i++) {
        int val = 0;
        if(pic[i]){ val = 255; }
        fprintf(fp, "%c%c%c", (unsigned char) val, (unsigned char) val, (unsigned char) val); 
    }
    fclose(fp);
}

__global__ void sobel(unsigned int *pic, bool *isEdge, int xsize, int ysize){
    int j = 3*(blockIdx.x*blockDim.x + threadIdx.x) + 1; // col
    int i = blockIdx.y*blockDim.y + threadIdx.y; // row

    int sum1, sum2, magnitude;
    int thresh = DEFAULT_THRESHOLD;

    if(i >= 1 && i < ysize-1 && j >= 1 && j < xsize*3-1){
        sum1 = pic[ (i-1)*xsize*3 + j+3 ] - pic[ (i-1)*xsize*3 + j-3 ] 
        + 2 * pic[ (i)*xsize*3 + j+3 ] - 2 * pic[ (i)*xsize*3 + j-3 ]
        + pic[ (i+1)*xsize*3 + j+3 ] - pic[ (i+1)*xsize*3 + j-3 ];
        
        sum2 = pic[ (i-1)*xsize*3 + j-3 ] + 2 * pic[ (i-1)*xsize*3 + j] + pic[ (i-1)*xsize*3 + j+3 ]
        - pic[ (i+1)*xsize*3 + j-3 ] - 2 * pic[ (i+1)*xsize*3 + j ] - pic[ (i+1)*xsize*3 + j+3 ];
        
        magnitude = sum1*sum1 + sum2*sum2;

        int offset = i*xsize + blockIdx.x*blockDim.x + threadIdx.x;
        if (magnitude > thresh){
            isEdge[offset] = true;
        }
        else {
            isEdge[offset] = false;
        } 
    }
}

__global__ void gcount_perpixel(unsigned int *pic, bool *isEdge, int *result, int xsize, int ysize, int *count, int startX, int startY){
    
    int col = startX + threadIdx.x*3;
    int row = startY + threadIdx.y;

    int offset = row*xsize*3 + col; // location of green value

    if( col < xsize*3 && row < ysize){
        int r = pic[offset];
        int g = pic[offset+1];
        int b = pic[offset+2];

        int thresh = 10;
        if(g-thresh > r && g-thresh > b){
            atomicAdd(count, 1);
            r = 140;
            b=g=0;
        }
        

        if(isEdge[row*xsize + col/3]){
            result[offset] = 255;
            result[offset+1] = 255;
            result[offset+2] = 255;
        } 
        else {
            result[offset] = r;
            result[offset+1] = g;
            result[offset+2] = b;
        }  

    }  

}

__global__ void gcount(unsigned int *pic, bool *isEdge, int *result, int xsize, int ysize, int *count){

    int cols = 16;
    int rows = 16;

    //TODO: I could speed this up by transposing the matrix and skipping unneeded rows

    int col = cols*3*(blockIdx.x*blockDim.x + threadIdx.x); // col
    int row = rows*(blockIdx.y*blockDim.y + threadIdx.y); // row

    // get average color
    float r = 0;
    float g = 0;
    float b = 0;

    int edgeCount = 0;

    for(int i = 0; i < cols*3; i+=3){
        for(int j = 0; j < rows; j++){
            int offset = (row+j)*xsize*3 + (col+i); // location of red value
    
            if( col < xsize*3 && row < ysize){
                r += pic[offset];
                g += pic[offset+1];
                b += pic[offset+2];

                edgeCount += (int)isEdge[(row+j)*xsize + (col/3+i)];

            }
        }
    }

    if(edgeCount > 25){

        dim3 grid(1, 1);
        dim3 block(16, 16);

        gcount_perpixel<<<grid, block>>>(pic, isEdge, result, xsize, ysize, count, col, row); 
    } else {
        r = r / (float)(cols*rows);
        g = g / (float)(cols*rows);
        b = b / (float)(cols*rows);
    
        int thresh = 10;
        if(g-thresh > r && g-thresh > b){
            atomicAdd(count, cols*rows);
            r=255;
            b=g=0;
        }

        for(int i = 0; i < cols; i++){
            for(int j = 0; j < rows; j++){
                int offset = (row+j)*xsize*3 + (col+i*3); // location of red value
        
                if( col < xsize*3 && row < ysize){
    
                    if(isEdge[(row+j)*xsize + (col/3+i)]){
                        result[offset] = 255;
                        result[offset+1] = 255;
                        result[offset+2] = 255;
                    } 
                    else {
                        result[offset] = (int)r;
                        result[offset+1] = (int)g;
                        result[offset+2] = (int)b;
                    }  
                }
            }
        }
    }

}

void checkCudaError(const char* task){
    hipError_t err = hipGetLastError();

    if (err != hipSuccess){
        fprintf(stderr, "Oops! (error code %s happened at \"%s\")!\n", hipGetErrorString(err), task); 
        exit(EXIT_FAILURE);
    }

    // fprintf(stderr, "Success! Completed \"%s\"!\n", task);
}

int main(int argc, char *argv[]){

    char* filename = strdup( argv[1] );
    int xsize, ysize, maxval;
    unsigned int *pic = read_ppm( filename, xsize, ysize, maxval); // define variables and read in image file

    // LOADING AND SETUP CODE ===================================================
    int numbytes =  xsize * ysize * 3 * sizeof( int ); // 3x because 3 floats for R, G, B channels
    int numbools =  xsize * ysize * sizeof( bool ); // edge detection boolean size

    hipEvent_t start_event, stop_event; // 
    float elapsed_time_par;
    
    unsigned int *d_pic = NULL; // pointer for device picture array

    bool *isEdge = (bool *) malloc( numbools ); // host and device edge boolean array
    bool *d_isEdge = NULL;

    int *result = (int *) malloc( numbytes ); // host and device result image array
    int *d_result = NULL;

    // SEQUENTIAL SOBEL ===================================================
    bool *seqIsEdge = (bool *) malloc( numbools );

    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);
    hipEventRecord(start_event, 0);

    for(int i = 0; i < ysize; i++){
        for (int j = 0; j < xsize; j++){
            int col = j*3 + 1;
            int sum1, sum2, magnitude;
            int thresh = DEFAULT_THRESHOLD;
        
            if(i >= 1 && i < ysize-1 && col >= 1 && col < xsize*3-1){
                sum1 = pic[ (i-1)*xsize*3 + col+3 ] - pic[ (i-1)*xsize*3 + col-3 ] 
                + 2 * pic[ (i)*xsize*3 + col+3 ] - 2 * pic[ (i)*xsize*3 + col-3 ]
                + pic[ (i+1)*xsize*3 + col+3 ] - pic[ (i+1)*xsize*3 + col-3 ];
                
                sum2 = pic[ (i-1)*xsize*3 + col-3 ] + 2 * pic[ (i-1)*xsize*3 + col] + pic[ (i-1)*xsize*3 + col+3 ]
                - pic[ (i+1)*xsize*3 + col-3 ] - 2 * pic[ (i+1)*xsize*3 + col ] - pic[ (i+1)*xsize*3 + col+3 ];
                
                magnitude = sum1*sum1 + sum2*sum2;
        
                int offset = i*xsize + j;
                if (magnitude > thresh){
                    seqIsEdge[offset] = true;
                }
                else {
                    seqIsEdge[offset] = false;
                } 
            }
        }
    }

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsed_time_par, start_event, stop_event);

    fprintf(stderr, "   Edge Detection Sequential Runtime: %f ms\n", elapsed_time_par);

    hipMalloc((void **) &d_pic, numbytes); // allocate input image space on device
    checkCudaError("allocate d_pic");

    hipMemcpy(d_pic, pic, xsize * ysize * sizeof(unsigned int) * 3 , hipMemcpyHostToDevice); // copy input image to device
    checkCudaError("copy d_pic");

    hipMalloc((void **) &d_isEdge, numbools); // allocate isEdge space on device
    checkCudaError("allocate d_isEdge");

    dim3 block(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 grid(ceil(xsize/ (float)BLOCK_SIZE_X ), ceil(ysize/ (float)BLOCK_SIZE_Y ));

    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);
    hipEventRecord(start_event, 0);

    // Launch edge detection kernel function
    // takes in pic array, returns boolean isEdge
    sobel<<<grid, block>>>(d_pic, d_isEdge, xsize, ysize); 
    checkCudaError("kernel launch");

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsed_time_par, start_event, stop_event);

    fprintf(stderr, "   Edge Detection Parallel Runtime: %f ms\n", elapsed_time_par);

    // GREEN PIXEL COUNTING CODE ================================================
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);
    hipEventRecord(start_event, 0);

    int count = 0;

    for(int i = 0; i < ysize; i++){
        for (int j = 0; j < xsize; j++){

            //TODO: I could speed this up by transposing the matrix and skipping unneeded rows

            int col = j*3; // col
            int row = i; // row

            // get average color
            float r = 0;
            float g = 0;
            float b = 0;

            for(int i = 0; i < 3; i+=3){
                for(int j = 0; j < 1; j++){
                    int offset = (row+j)*xsize*3 + (col+i); // location of red value
            
                    if( col < xsize*3 && row < ysize){
                        r += pic[offset];
                        g += pic[offset+1];
                        b += pic[offset+2];
                    }
                }
            }

            int thresh = 10;
            if(g-thresh > r && g-thresh > b){
                count++;
                r=255;
                b=g=0;
            }

            for(int i = 0; i < 1; i++){
                for(int j = 0; j < 1; j++){
                    int offset = (row+j)*xsize*3 + (col+i*3); // location of red value
            
                    if( col < xsize*3 && row < ysize){

                        if(isEdge[(row+j)*xsize + (col/3+i)]){
                            result[offset] = 255;
                            result[offset+1] = 255;
                            result[offset+2] = 255;
                        } 
                        else {
                            result[offset] = (int)r;
                            result[offset+1] = (int)g;
                            result[offset+2] = (int)b;
                        }  
                    }
                }
            }
        }
    }

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsed_time_par, start_event, stop_event);

    fprintf(stderr, "   Pixel Counting Sequential Runtime: %f ms\n", elapsed_time_par);

    hipMalloc((void **) &d_result, numbytes); // allocate result image space on device
    checkCudaError("allocate d_result");

    dim3 grid2(ceil((xsize/16)/ (float)BLOCK_SIZE_X ), ceil((ysize/16)/ (float)BLOCK_SIZE_Y ));

    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);
    hipEventRecord(start_event, 0);

    int *num_pix_found;
    hipMallocManaged(&num_pix_found, 4); // allocate space for num_pix_found on device
    *num_pix_found = 0;

    // Launch pixel count kernel function
    // takes in input pic array and boolean isEdge, returns num_pix_found and result image array
    gcount<<<grid2, block>>>(d_pic, d_isEdge, d_result, xsize, ysize, num_pix_found); 
    checkCudaError("kernel launch");

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsed_time_par, start_event, stop_event);

    fprintf(stderr, "   Pixel Counting Parallel Runtime: %f ms\n", elapsed_time_par);

    // fprintf(stderr, "Count Runtime: %f ms\n", elapsed_time_par);

    hipMemcpy(result, d_result, numbytes, hipMemcpyDeviceToHost); // copy result image to host
    checkCudaError("copy d_result");

    fprintf(stderr, "   file: %s, num_pix_found: %d, cm^2: %d\n",filename, *num_pix_found, *num_pix_found / 467); // there are 466.667 pixels per cm^2

    hipFree(d_pic);
    hipFree(d_isEdge);
    hipFree(d_result);

    write_ppm("sobel.ppm", xsize, ysize, 255, result); // write result image file
}
